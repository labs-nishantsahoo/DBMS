#include "hip/hip_runtime.h"

#include <stdlib.h> //for malloc
#include <stdio.h> //for pf and sf

#define BLOCK_DIM 16
#define SCALE_FACTOR 2

__device__ float  *dPixelIn, *dPixelOut;

//const used for pixelIn just to avoid accidental change in kernel code
__global__ void scalePicture(int M, int N)
{
    int row = blockIdx.y* blockDim.y + threadIdx.y; //row is in y direction
    int col = blockIdx.x* blockDim.x + threadIdx.x; //column in x direction
    if( row < M && col < N)
        // each row contains N elements, so skip it for each row to get ith row , jth column element
        pixelOut[row *N + col]=SCALE_FACTOR *pixelIn[row *N + col];
}

int main(void)
{

  float *pixelIn, *pixelOut;

  //Variables to allocate in GPU memory
  
  int M, N, i,j;
  printf("Enter dimension of picture\n");
  scanf("%d%d", &M, &N);
  //allocate in CPU memory
  int size=sizeof(float)*M*N;
  pixelIn=(float *)malloc(size);
  pixelOut=(float*) malloc(size);
  
  //read image pixels from  image file
  // time being using some random values;
  for(i=0; i<M; i++){
   for(j=0; j<N; j++){
     pixelIn[i*N+j]= i*j+1;
     printf("%f ", pixelIn[i*N+j]);}
   printf("\n");
}
    
  hipMalloc(&dPixelIn,size);
  hipMalloc(&dPixelOut,size);

  //copy original pixels to GPU
  hipMemcpy(dPixelIn,pixelIn,size, hipMemcpyHostToDevice);
  
  //Launch Kernel along x direction as many (columns/ threads per block)
  //Along y direction as many as rows/threads per block
  dim3 gridSize(ceil(N/(float) BLOCK_DIM), ceil(M/(float)BLOCK_DIM )), blockSize(BLOCK_DIM,BLOCK_DIM);
  
  scalePicture<<<gridSize,blockSize >>>( M, N);
  hipMemcpy(pixelOut,dPixelOut,size, hipMemcpyDeviceToHost);
  
     
  printf("New scaled picture with scale factor %d  is \n", SCALE_FACTOR);

  for(i=0; i<M; i++){
   for(j=0; j<N; j++)
     printf("%f ", pixelOut[i*N+j]);
   printf("\n");
   }
   free(pixelIn);
   free(pixelOut);
   hipFree(dPixelIn);
   hipFree(dPixelOut);
   return 0;
}

