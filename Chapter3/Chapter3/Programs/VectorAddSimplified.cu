
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cstdio>
#include <math.h>
// this kernel computes the vector sum c = a + b
// each thread performs one pair-wise addition
__global__ void vector_add(const float *a,
                           const float *b,
                           float *c,
                           const size_t n){
 unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
  // avoid accessing out of bounds elements
  if(i < n)
  {
    // sum elements
    c[i] = a[i] + b[i];
  }
}

int main(void)
{
  // create arrays of 1M elements
  int num_elements = 0 ;
  printf("Enter number of elements to add");
  scanf("%d", &num_elements);
  // compute the size of the arrays in bytes
  const int num_bytes = num_elements * sizeof(float);

  // points to host & device arrays
  float *array_a;
  float *array_b;
  float *array_c;
  // cudaMalloc the device arrays
 hipMallocManaged((void**)&array_a, num_bytes);
 hipMallocManaged((void**)&array_b, num_bytes);
 hipMallocManaged((void**)&array_c, num_bytes);

  // initialize host_array_a & host_array_b
  for(int i = 0; i < num_elements; ++i)
  {
    // make array a a linear ramp
    array_a[i] = (float)i;

    // make array b random
    array_b[i] = (float)rand() / RAND_MAX;
  }
    
  
  vector_add <<< ceil(num_elements/32.0), 32>>>
(array_a, array_b, array_c, num_elements);
hipDeviceSynchronize();
  for(int i = 0; i < num_elements; ++i)
  {
    printf("result %d: %1.1f + %7.1f = %7.1f\n", i, array_a[i], array_b[i], array_c[i]);
  }

  hipFree(array_a);
  hipFree(array_b);
  hipFree(array_c);
}

