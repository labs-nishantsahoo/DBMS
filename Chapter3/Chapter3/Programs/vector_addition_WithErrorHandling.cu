
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError( hipError_t err, const char *file, int line ) 
{
       if (err != hipSuccess) 
    {
	 printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
	 exit( EXIT_FAILURE );
    }
}

// this kernel computes the vector sum c = a + b
// each thread performs one pair-wise addition
__global__ void vector_add(const float *a,
                           const float *b,
                           float *c,
                           const size_t n)
{
  // compute the global element index this thread should process
  unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

  // avoid accessing out of bounds elements
  if(i < n)
  {
    // sum elements
    c[i] = a[i] + b[i];
  }
}



int main(void)
{
  // create arrays of 1M elements
  int num_elements = 0 ;
  printf("Enter number of elements to add");
  scanf("%d", &num_elements);
  // compute the size of the arrays in bytes
  const int num_bytes = num_elements * sizeof(float);

  // points to host & device arrays
  float *device_array_a = 0;
  float *device_array_b = 0;
  float *device_array_c = 0;

  float *host_array_a   = 0;
  float *host_array_b   = 0;
  float *host_array_c   = 0;

  // malloc the host arrays
  host_array_a = (float*)malloc(num_bytes);
  host_array_b = (float*)malloc(num_bytes);
  host_array_c = (float*)malloc(num_bytes);

  // cudaMalloc the device arrays
  HANDLE_ERROR(hipMalloc((void**)&device_array_a, num_bytes));
  HANDLE_ERROR(hipMalloc((void**)&device_array_b, num_bytes));
  HANDLE_ERROR(hipMalloc((void**)&device_array_c, num_bytes));


  // initialize host_array_a & host_array_b
  for(int i = 0; i < num_elements; ++i)
  {
    // make array a a linear ramp
    host_array_a[i] = (float)i;

    // make array b random
    host_array_b[i] = (float)rand() / RAND_MAX;
  }

 
       
  // copy arrays a & b to the device memory space
  HANDLE_ERROR(hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice));



  // launch the kernel
    
  vector_add <<< ceil(num_elements/32.0), 32>>>(device_array_a, device_array_b, device_array_c, num_elements);

  // copy the result back to the host memory space
  HANDLE_ERROR(hipMemcpy(host_array_c, device_array_c, num_bytes, hipMemcpyDeviceToHost));

  for(int i = 0; i < num_elements; ++i)
  {
    printf("result %d: %1.1f + %7.1f = %7.1f\n", i, host_array_a[i], host_array_b[i], host_array_c[i]);
  }

  // deallocate memory
  free(host_array_a);
  free(host_array_b);
  free(host_array_c);

  HANDLE_ERROR(hipFree(device_array_a));
  HANDLE_ERROR(hipFree(device_array_b));
  HANDLE_ERROR(hipFree(device_array_c));
}

